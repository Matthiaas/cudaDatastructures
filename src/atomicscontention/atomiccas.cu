#include "hip/hip_runtime.h"
#include "atomiccas.cuh"

#include <iostream>
#include <math.h>

#include "hip/hip_runtime.h"
#include "" 
#include <hipcub/hipcub.hpp>

#define WARPSIZE 32
#define BLOCKSIZE 1024

namespace atomiccas {

__global__
void add_as_accumuluated_requests(int *v)
{
  const int warp_count = BLOCKSIZE / WARPSIZE;
  int warp_id = threadIdx.x / WARPSIZE;

  typedef hipcub::WarpScan<int> WarpScan;
  __shared__ typename WarpScan::TempStorage temp_storage[warp_count];

  __shared__ int requests[warp_count * WARPSIZE];

  int value = 1;
  int pos;

  WarpScan(temp_storage[warp_id]).ExclusiveSum(value, pos);

  requests[warp_id * WARPSIZE + pos] = value;
  __syncwarp();
  if(threadIdx.x % WARPSIZE == 0) {
    int sum = 0;
    for (int i = 0; i < WARPSIZE; i++) {
      sum += requests[warp_id * WARPSIZE + i];
    }

    while(1) {
      int val = *v;
      if (atomicCAS(v, val, val + sum) == val) {
        break;
      }
    }
  }
  
}



__global__
void add_as_requests(int *v)
{
  const int warp_count = BLOCKSIZE / WARPSIZE;
  int warp_id = threadIdx.x / WARPSIZE;

  typedef hipcub::WarpScan<int> WarpScan;
  __shared__ typename WarpScan::TempStorage temp_storage[warp_count];

  __shared__ int requests[warp_count * WARPSIZE];

  int value = 1;
  int pos;

  WarpScan(temp_storage[warp_id]).ExclusiveSum(value, pos);

  requests[warp_id * WARPSIZE + pos] = value;
  __syncwarp();
  if(threadIdx.x % WARPSIZE == 0) {
    for (int i = 0; i < WARPSIZE; i++) {
      
      while(1) {
        int val = *v;
        if (atomicCAS(v, val, val + requests[warp_id * WARPSIZE + i]) == val) {
          break;
        }
      }
    }
  }
  
}

__global__
void add_trival(int *v)
{
  while(1) {
    int val = *v;
    if (atomicCAS(v, val, val + 1) == val) {
      break;
    }
  }
}

}

// int main(void)
// {
//   int *v;
//   hipMalloc(&v, sizeof(int));


  
//   int blocks = 1024 * 4;
//   int threads = BLOCKSIZE;

//   auto map = std::map<std::string, std::function<void(void)>>{
//     {"cadd_as_accumuluated_requests", [&] { add_as_accumuluated_requests<<<blocks, threads>>>(v); }},
//     {"badd_as_requests", [&] { add_as_requests<<<blocks, threads>>>(v); }},
//     {"aadd_trival", [&] { add_trival<<<blocks, threads>>>(v); }},
//   };

//   auto init = [&] { hipMemset(v, 0, sizeof(int)); };

//   auto validate = [&] {
//     int h_v;
//     hipMemcpy(&h_v, v, sizeof(int), hipMemcpyDeviceToHost);
//     return (h_v == (blocks * threads));
//   };
  
//   timeKernels(init, map, validate);
  

//   // Free memory
//   hipFree(v);
  
//   return 0;
// }